#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matrixVectorMul(double *A, double *p, double *Ap, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n) {
        double sum = 0.0;
        for (int col = 0; col < n; col++) {
            sum += A[row * n + col] * p[col];
        }
        Ap[row] = sum;
    }
}

__global__ void dotProduct(double *a, double *b, double *result, int n) {
    extern __shared__ double cache[];
    int tid = threadIdx.x;
    cache[tid] = 0.0;
    if (tid < n) cache[tid] = a[tid] * b[tid];
    __syncthreads();

    // Reduction
    for (int i = blockDim.x/2; i > 0; i /= 2) {
        if (tid < i && tid + i < n) {
            cache[tid] += cache[tid + i];
        }
        __syncthreads();
    }
    if (tid == 0) atomicAdd(result, cache[0]);
}

void conjugateGradientCUDA(double *A, double *b, double *x, int n, double tol=1e-6) {
    // Allocate device memory
    double *d_A, *d_b, *d_x, *d_r, *d_p, *d_Ap;
    hipMalloc(&d_A, n*n*sizeof(double));
    hipMalloc(&d_b, n*sizeof(double));
    hipMalloc(&d_x, n*sizeof(double));
    hipMalloc(&d_r, n*sizeof(double));
    hipMalloc(&d_p, n*sizeof(double));
    hipMalloc(&d_Ap, n*sizeof(double));

    // Copy data to device
    hipMemcpy(d_A, A, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_x, 0, n*sizeof(double));

    // Initialize residuals
    hipMemcpy(d_r, d_b, n*sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(d_p, d_r, n*sizeof(double), hipMemcpyDeviceToDevice);

    double rsq_old, alpha, beta;
    double *d_rsq_old, *d_alpha, *d_beta;
    hipMalloc(&d_rsq_old, sizeof(double));
    hipMalloc(&d_alpha, sizeof(double));
    hipMalloc(&d_beta, sizeof(double));

    // Compute initial residual squared
    dotProduct<<<1, 256, 256*sizeof(double)>>>(d_r, d_r, d_rsq_old, n);

    for (int k = 0; k < 1000; ++k) {
        // Compute Ap = A * p
        matrixVectorMul<<<(n+255)/256, 256>>>(d_A, d_p, d_Ap, n);

        // Compute alpha
        double pAp;
        dotProduct<<<1, 256, 256*sizeof(double)>>>(d_p, d_Ap, d_alpha, n);
        hipMemcpy(&alpha, d_alpha, sizeof(double), hipMemcpyDeviceToHost);
        alpha = rsq_old / alpha;

        // Update x and r
        hipblasDaxpy(n, alpha, d_p, 1, d_x, 1);
        hipblasDaxpy(n, -alpha, d_Ap, 1, d_r, 1);

        // Check convergence
        double rsq_new;
        dotProduct<<<1, 256, 256*sizeof(double)>>>(d_r, d_r, &rsq_new, n);
        if (sqrt(rsq_new) < tol) break;

        // Update beta and p
        beta = rsq_new / rsq_old;
        hipblasDscal(n, beta, d_p, 1);
        hipblasDaxpy(n, 1.0, d_r, 1, d_p, 1);
        rsq_old = rsq_new;
    }

    // Copy result back
    hipMemcpy(x, d_x, n*sizeof(double), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A); hipFree(d_b); hipFree(d_x);
    hipFree(d_r); hipFree(d_p); hipFree(d_Ap);
}
